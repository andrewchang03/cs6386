
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

#include <hip/hip_runtime_api.h>
#include <hipsparse.h>

#include <stdio.h>  // printf
#include <stdlib.h> // EXIT_FAILURE

#define CHECK_CUDA(func)                                               \
    {                                                                  \
        hipError_t status = (func);                                   \
        if (status != hipSuccess)                                     \
        {                                                              \
            printf("CUDA API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipGetErrorString(status), status);      \
            return EXIT_FAILURE;                                       \
        }                                                              \
    }

#define CHECK_CUSPARSE(func)                                               \
    {                                                                      \
        hipsparseStatus_t status = (func);                                  \
        if (status != HIPSPARSE_STATUS_SUCCESS)                             \
        {                                                                  \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
                   __LINE__, hipsparseGetErrorString(status), status);      \
            return EXIT_FAILURE;                                           \
        }                                                                  \
    }

int *hRowPtrs, *hColInds, *dRowPtrs, *dColInds;
float *hData, *dData;

template <typename T>
void print_from_device(T *dBuffer, size_t count)
{
    T *hBuffer = (T *)malloc(count * sizeof(T));
    hipMemcpy(hBuffer, dBuffer, count * sizeof(T), hipMemcpyDeviceToHost);

    for (int i = 0; i < count; ++i)
        std::cout << hBuffer[i] << " ";
    std::cout << std::endl;

    free(hBuffer);
}

void read_data(std::string fName, size_t nrows, size_t nnz)
{
    std::string path = "data/" + fName + "/" + fName + "-";

    // init files
    std::ifstream fRowPtrs(path + "rowptrs", std::ios::binary);
    std::ifstream fColInds(path + "colinds", std::ios::binary);
    std::ifstream fData(path + "data", std::ios::binary);

    // read files into host memory
    fRowPtrs.read(reinterpret_cast<char *>(hRowPtrs), (nrows + 1) * sizeof(int));
    fColInds.read(reinterpret_cast<char *>(hColInds), nnz * sizeof(int));
    fData.read(reinterpret_cast<char *>(hData), nnz * sizeof(float));

    fRowPtrs.close();
    fColInds.close();
    fData.close();

    // for (size_t i = 0; i < nnz; ++i) {
    //     std::cout << hData[i] << " ";
    // }
    // std::cout << std::endl;
}

int main(int argc, char *argv[])
{
    // READ DATA
    char *fName = argv[1];
    int nrows = std::atoi(argv[2]);
    int ncols = std::atoi(argv[3]);
    int nnz = std::atoi(argv[4]);

    hRowPtrs = (int *)calloc(nrows + 1, sizeof(int));
    hColInds = (int *)calloc(nnz, sizeof(int));
    hData = (float *)calloc(nnz, sizeof(float));

    read_data(fName, nrows, nnz);

    CHECK_CUDA(hipMalloc((void **)&dRowPtrs, (nrows + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&dColInds, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&dData, nnz * sizeof(float)))

    CHECK_CUDA(hipMemcpy(dRowPtrs, hRowPtrs, (nrows + 1) * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dColInds, hColInds, nnz * sizeof(int),
                          hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dData, hData, nnz * sizeof(float),
                          hipMemcpyHostToDevice))

    // CREATE SPARSE MATRICES U (matA), UT (matB)
    hipsparseHandle_t handle = NULL;
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    hipsparseSpMatDescr_t matA, matB, matC; // matC is the product AB

    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, nrows, ncols, nnz,
                                     dRowPtrs, dColInds, dData,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))

    // TRANSPOSE U (matA)
    int *dCscColOffsets, *dCscRowInds;
    float *dCscData;
    CHECK_CUDA(hipMalloc(&dCscColOffsets, (ncols + 1) * sizeof(int)))
    CHECK_CUDA(hipMalloc(&dCscRowInds, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc(&dCscData, nnz * sizeof(float)))
    size_t cscBufferSize = 0;
    void *dCscBuffer = nullptr;
    CHECK_CUSPARSE(hipsparseCsr2cscEx2_bufferSize(
        handle, nrows, ncols, nnz,
        dData, dRowPtrs, dColInds,
        dCscData, dCscColOffsets, dCscRowInds,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        &cscBufferSize))
    CHECK_CUDA(hipMalloc(&dCscBuffer, cscBufferSize))
    CHECK_CUSPARSE(hipsparseCsr2cscEx2(
        handle, nrows, ncols, nnz,
        dData, dRowPtrs, dColInds,
        dCscData, dCscColOffsets, dCscRowInds,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        dCscBuffer))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matB, ncols, nrows, nnz,
                                     dCscColOffsets, dCscRowInds, dCscData,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))

    int *dSpgemmRowPtrs, *dSpgemmColInds;
    float *dSpgemmData;
    CHECK_CUDA(hipMalloc((void **)&dSpgemmRowPtrs,
                          (nrows + 1) * sizeof(int)))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matC, nrows, nrows, 0,
                                     dSpgemmRowPtrs, NULL, NULL,
                                     HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                     HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F))

    // SPGEMM COMPUTATION AB (U * UT)
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType computeType = HIP_R_32F;
    void *dSpgemmBuffer1 = NULL, *dSpgemmBuffer2 = NULL;
    size_t spgemmBufferSize1 = 0, spgemmBufferSize2 = 0;

    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE(hipsparseSpGEMM_createDescr(&spgemmDesc))

    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_ALG2,
                                      spgemmDesc, &spgemmBufferSize1, NULL))
    CHECK_CUDA(hipMalloc((void **)&dSpgemmBuffer1, spgemmBufferSize1))
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, matA, matB, &beta, matC,
                                      computeType, HIPSPARSE_SPGEMM_ALG2,
                                      spgemmDesc, &spgemmBufferSize1, dSpgemmBuffer1))
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, matA, matB, &beta, matC,
                               computeType, HIPSPARSE_SPGEMM_ALG2,
                               spgemmDesc, &spgemmBufferSize2, NULL))
    CHECK_CUDA(hipMalloc((void **)&dSpgemmBuffer2, spgemmBufferSize2))
    CHECK_CUSPARSE(hipsparseSpGEMM_compute(handle, opA, opB,
                                          &alpha, matA, matB, &beta, matC,
                                          computeType, HIPSPARSE_SPGEMM_ALG2,
                                          spgemmDesc, &spgemmBufferSize2, dSpgemmBuffer2))

    int64_t spgemmNrows, spgemmNcols, spgemmNnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matC, &spgemmNrows, &spgemmNcols,
                                        &spgemmNnz))
    CHECK_CUDA(hipMalloc((void **)&dSpgemmColInds, spgemmNnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void **)&dSpgemmData, spgemmNnz * sizeof(float)))
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dSpgemmRowPtrs, dSpgemmColInds, dSpgemmData))
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_ALG2, spgemmDesc))

    std::cout << spgemmNrows << " " << spgemmNcols << std::endl;
    // print_from_device(dSpgemmData, spgemmNnz);

    // DESTROY
    CHECK_CUSPARSE(hipsparseSpGEMM_destroyDescr(spgemmDesc))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matC))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

    CHECK_CUDA(hipFree(dSpgemmBuffer1))
    CHECK_CUDA(hipFree(dSpgemmBuffer2))
    CHECK_CUDA(hipFree(dRowPtrs))
    CHECK_CUDA(hipFree(dColInds))
    CHECK_CUDA(hipFree(dData))
    CHECK_CUDA(hipFree(dCscColOffsets))
    CHECK_CUDA(hipFree(dCscRowInds))
    CHECK_CUDA(hipFree(dCscData))
    CHECK_CUDA(hipFree(dSpgemmRowPtrs))
    CHECK_CUDA(hipFree(dSpgemmColInds))
    CHECK_CUDA(hipFree(dSpgemmData))

    free(hRowPtrs);
    free(hColInds);
    free(hData);

    return 0;
}